#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "ArduinoJson.h"

#define devij(dimx, dimy) \
int i = blockIdx.x % dimx; \
int j = threadIdx.x + (blockIdx.x - i) / dimx * dimy / d_nbt; \
int ij = i * dimy + j

typedef struct{
    int nx;
    int nz;
    int nt;
    float dt;
    float Lx;
    float Lz;
} fwdcfg;
typedef struct{
    float *stf_x;
    float *stf_y;
    float *stf_z;
} fwdarg;
typedef struct{
    float *vx;
    float *vz;
} fwddat;

const int nbt = 8;
__constant__ int d_nbt = 8;

namespace mat{
    float *create(const int m) {
    	float *a;
    	hipMalloc((void**)&a, m * sizeof(float));
    	return a;
    }
    float *create_h(const int m) {
    	return (float *)malloc(m * sizeof(float));
    }
    int *create_i(const int m){
        int *a;
    	hipMalloc((void**)&a, m * sizeof(int));
    	return a;
    }

    __global__ void set_d(float *a, const float init, const int m, const int n){
        devij(m, n);
        a[ij] = init;
    }
    void set(float *a, const float init, const int m, const int n){
        mat::set_d<<<m * nbt, n / nbt>>>(a, init, m, n);
    }
    void copyhd(float *d_a, const float *a, const int m){
        hipMemcpy(d_a, a , m * sizeof(float), hipMemcpyHostToDevice);
    }
    void copydh(float *a, const float *d_a, const int m){
        hipMemcpy(a, d_a , m * sizeof(float), hipMemcpyDeviceToHost);
    }
    void write(FILE *file, float *d_a, float *a, const int m){
        mat::copydh(a, d_a, m);
        fwrite(a, sizeof(float), m, file);
    }
    void read(FILE *file, float *a, const int m){
        fread(a, sizeof(float), m, file);
    }
}

fwdcfg import_data(void){
    fwdcfg cfg;
    FILE *cfgfile = fopen("externaltools/config","r");

    char *buffer = 0;
    long length;

    fseek (cfgfile, 0, SEEK_END);
    length = ftell (cfgfile);
    fseek (cfgfile, 0, SEEK_SET);
    buffer = (char *)malloc (length + 1);
    fread (buffer, 1, length, cfgfile);
    buffer[length] = '\0';

    fclose(cfgfile);

    if (buffer){
        DynamicJsonBuffer jsonBuffer;
        JsonObject& root = jsonBuffer.parseObject(buffer);
        if (!root.success()){
            printf("parseObject() failed\n");
        }
        else{
            cfg.nx = root["nx"];
            cfg.nz = root["nz"];
            cfg.nt = root["nt"];
            cfg.dt = root["dt"];
            cfg.Lx = root["Lx"];
            cfg.Lz = root["Lz"];
        }
    }
    return cfg;
}
float *import_data(char *path, int *len){
    char fpath[50] = "externaltools/";
    strcat(fpath, path);
    *len = 0;
    float *data = 0;
    FILE *datafile = fopen(fpath,"r");
    if(datafile){
        while(!feof(datafile)){
            float datavalue;
            fscanf(datafile, "%f\n", &datavalue);
            *len = *len + 1;
        }
        fclose(datafile);

        datafile = fopen(fpath,"r");
        data = mat::create_h(*len);
        for(int i=0; i<*len; i++){
            fscanf(datafile, "%f\n", data + i);
        }
        fclose(datafile);
    }
    return data;
}
void define_computational_domain(float Lx, float Lz, int nx, int nz, float *dx, float *dz){
    *dx = Lx / (nx - 1);
    *dz = Lz / (nz - 1);
}
fwdarg prepare_stf(fwdcfg cfg){
    float dx, dz;
    define_computational_domain(cfg.Lx, cfg.Lz, cfg.nx, cfg.nz, &dx, &dz);
    float *t = mat::create_h(cfg.nt);
    for(int i = 0; i < cfg.nt; i++){
        t[i] = i * cfg.dt;
    }
    fwdarg stf;// from here
    // float *data = mat::create_h(cfg.nt);
    // for(int i=0;i<cfg.nt;i++){
    //     data[i]=i;
    //     // from here
    // }
    // stf.stf_x = data;
    printf("lxz %f %f\n",dx,dz);
    return stf;
}
fwdarg checkstf(fwdcfg cfg){
    int len;
    float *stfall = import_data("stf", &len);
    if(len > 0){
        fwdarg stf;
        stf.stf_x = stfall;
        return stf;
    }
    else{
        return prepare_stf(cfg);
    }
}
void run_wavefield_propagation(void){

}
void run_forward(void){
    fwdcfg cfg = import_data();
    fwdarg stf = checkstf(cfg);
    printf("nx: %d\nnt: %d\n", cfg.nx, cfg.nt);
    printf("stf: %f %f %f\n",stf.stf_x[0],stf.stf_x[1],stf.stf_x[2]);
}

int main(int argc , char *argv[]){
    for(int i = 0; i< argc; i++){
        if(strcmp(argv[i],"run_forward") == 0){
            run_forward();
        }
    }
    return 0;
}
