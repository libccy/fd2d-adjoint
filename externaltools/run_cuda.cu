#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "ArduinoJson.h"

#define devij(dimx, dimy) \
int i = blockIdx.x % dimx; \
int j = threadIdx.x + (blockIdx.x - i) / dimx * dimy / d_nbt;

const float pi = 3.1415927;
const int nbt = 8;
__constant__ int d_nbt = 8;

typedef struct{
    int nx;
    int nz;
    int nt;
    float dx;
    float dz;
    float dt;
    float Lx;
    float Lz;

    int sfe;
    int update_params;
    int model_type;
    float source_amplitude;

    int nsrc;
    int nrec;
    int *stf_type;
    float *src_x;
    float *src_z;
    float *stf_PSV_x;
    float *stf_PSV_z;
    float *tauw_0;
    float *tauw;
    float *tee_0;
    float *f_min;
    float *f_max;
    float *rec_x;
    float *rec_z;

    int **src_x_id;
    int **src_z_id;
    int **rec_x_id;
    int **rec_z_id;

    float **stf_x;
    float **stf_y;
    float **stf_z;

    float **lambda;
    float **mu;
    float **rho;
} fdat;

namespace mat{
    __global__ void _setValue(float **mat, const float init, const int m, const int n){
        devij(m, n);
        mat[i][j] = init;
    }
    __global__ void _setValue(float *mat, const float init, const int m){
        int i = threadIdx.x;
        mat[i] = init;
    }
    __global__ void _setPointerValue(float **mat, float *data, const int n){
        int i = threadIdx.x;
        mat[i] = data + n * i;
    }
    __global__ void _setIntPointerValue(int **mat, int *data, const int n){
        int i = threadIdx.x;
        mat[i] = data + n * i;
    }


    float *init(float *mat, const int m, const float init){
        mat::_setValue<<<1, m>>>(mat, init, m);
        return mat;
    }
    float **init(float **mat, const int m, const int n, const float init){
        mat::_setValue<<<m * nbt, n / nbt>>>(mat, init, m, n);
        return mat;
    }
    float *initHost(float *mat, const int m, const float init){
        for(int i = 0; i < m; i++){
            mat[i] = init;
        }
        return mat;
    }
    float **initHost(float **mat, const int m, const int n, const float init){
        for(int i = 0; i < m; i++){
            for(int j = 0; j < n; j++){
                mat[i][j] = init;
            }
        }
        return mat;
    }

    float *create(const int m) {
    	float *data;
    	hipMalloc((void**)&data, m * sizeof(float));
    	return data;
    }
    float **create(const int m, const int n){
    	float *data;
    	hipMalloc((void**)&data, m * n * sizeof(float));
        float **mat;
        hipMalloc((void**)&mat, m * sizeof(float *));
        mat::_setPointerValue<<<1, m>>>(mat, data, n);
    	return mat;
    }
    float **create(const int m, const int n, const float init){
        return mat::init(mat::create(m,n), m, n, init);
    }
    float *createHost(const int m) {
    	return (float *)malloc(m * sizeof(float));
    }
    float **createHost(const int m, const int n){
    	float **mat = (float **)malloc(m * sizeof(float *));
    	float *data = (float *)malloc(m * n * sizeof(float));
    	for(int i  =0; i < m; i++){
    		mat[i] = data + n * i;
    	}
    	return mat;
    }
    float **createHost(const int m, const int n, const float init){
        return mat::initHost(mat::createHost(m,n), m, n, init);
    }
    int *createInt(const int m){
        int *a;
    	hipMalloc((void**)&a, m * sizeof(int));
    	return a;
    }
    int **createInt(const int m, const int n){
    	int *data;
    	hipMalloc((void**)&data, m * n * sizeof(int));
        int **mat;
        hipMalloc((void**)&mat, m * sizeof(int *));
        mat::_setIntPointerValue<<<1, m>>>(mat, data, n);
    	return mat;
    }
    int *createIntHost(const int m) {
    	return (int *)malloc(m * sizeof(int));
    }
    int **createIntHost(const int m, const int n){
    	int **mat = (int **)malloc(m * sizeof(int *));
    	int *data = (int *)malloc(m * n * sizeof(int));
    	for(int i  =0; i < m; i++){
    		mat[i] = data + n * i;
    	}
    	return mat;
    }


    void copyHostToDevice(float *d_a, const float *a, const int m){
        hipMemcpy(d_a, a , m * sizeof(float), hipMemcpyHostToDevice);
    }
    void copyHostToDevice(float **pd_a, float **pa, const int m, const int n){
        float **phd_a=(float **)malloc(sizeof(float *));
        hipMemcpy(phd_a, pd_a , sizeof(float *), hipMemcpyDeviceToHost);
        hipMemcpy(*phd_a, *pa , m * n * sizeof(float), hipMemcpyHostToDevice);
    }
    void copyDeviceToHost(float *a, const float *d_a, const int m){
        hipMemcpy(a, d_a , m * sizeof(float), hipMemcpyDeviceToHost);
    }
    void copyDeviceToHost(float **pa, float **pd_a, const int m, const int n){
        float **phd_a=(float **)malloc(sizeof(float *));
        hipMemcpy(phd_a, pd_a , sizeof(float *), hipMemcpyDeviceToHost);
        hipMemcpy(*pa, *phd_a , m * n * sizeof(float), hipMemcpyDeviceToHost);
    }
}

fdat *importData(void){
    fdat *dat = new fdat;
    FILE *datfile = fopen("externaltools/config","r");

    char *buffer = 0;
    long length;

    fseek (datfile, 0, SEEK_END);
    length = ftell (datfile);
    fseek (datfile, 0, SEEK_SET);
    buffer = (char *)malloc (length + 1);
    fread (buffer, 1, length, datfile);
    buffer[length] = '\0';

    fclose(datfile);

    if (buffer){
        DynamicJsonBuffer jsonBuffer;
        JsonObject& root = jsonBuffer.parseObject(buffer);
        if (!root.success()){
            printf("parseObject() failed\n");
        }
        else{
            dat->nx = root["nx"];
            dat->nz = root["nz"];
            dat->nt = root["nt"];
            dat->dt = root["dt"];
            dat->Lx = root["Lx"];
            dat->Lz = root["Lz"];

            dat->sfe = root["sfe"];
            dat->model_type = root["model_type"];
            dat->source_amplitude = root["source_amplitude"];

            int single_src = root["src_info"].is<JsonObject>();
            int single_rec = root["rec_x"].is<float>();

            dat->nsrc = single_src?1:root["src_info"].size();
            dat->stf_type = mat::createIntHost(dat->nsrc);
            dat->src_x = mat::createHost(dat->nsrc);
            dat->src_z = mat::createHost(dat->nsrc);
            dat->stf_PSV_x = mat::createHost(dat->nsrc);
            dat->stf_PSV_z = mat::createHost(dat->nsrc);
            dat->tauw_0 = mat::createHost(dat->nsrc);
            dat->tauw = mat::createHost(dat->nsrc);
            dat->tee_0 = mat::createHost(dat->nsrc);
            dat->f_min = mat::createHost(dat->nsrc);
            dat->f_max = mat::createHost(dat->nsrc);

            if(single_src){
                JsonObject& src = root["src_info"];
                dat->src_x[0] = src["loc_x"];
                dat->src_z[0] = src["loc_z"];
                dat->stf_type[0] = 2;
                dat->stf_PSV_x[0] = src["stf_PSV"][0];
                dat->stf_PSV_z[0] = src["stf_PSV"][1];
                dat->tauw_0[0] = src["tauw_0"];
                dat->tauw[0] = src["tauw"];
                dat->tee_0[0] = src["tee_0"];
                dat->f_min[0] = src["f_min"];
                dat->f_max[0] = src["f_max"];
            }
            else{
                JsonArray& src_info = root["src_info"];
                for(int i = 0; i < dat->nsrc; i++){
                    JsonObject& src = src_info.get<JsonObject>(i);
                    dat->src_x[i] = src["loc_x"];
                    dat->src_z[i] = src["loc_z"];
                    dat->stf_type[i] = 2; // ricker: modify later
                    dat->stf_PSV_x[i] = src["stf_PSV"][0];
                    dat->stf_PSV_z[i] = src["stf_PSV"][1];
                    dat->tauw_0[i] = src["tauw_0"];
                    dat->tauw[i] = src["tauw"];
                    dat->tee_0[i] = src["tee_0"];
                    dat->f_min[i] = src["f_min"];
                    dat->f_max[i] = src["f_max"];
                }
            }

            dat->nrec = single_rec?1:root["rec_x"].size();
            dat->rec_x = mat::createHost(dat->nrec);
            dat->rec_z = mat::createHost(dat->nrec);
            
            if(single_rec){
                dat->rec_x[0] = root["rec_x"];
                dat->rec_z[0] = root["rec_z"];
            }
            else{
                JsonArray& rec_x = root["rec_x"];
                JsonArray& rec_z = root["rec_z"];
                for(int i = 0; i < dat->nrec; i++){
                    dat->rec_x[i] = rec_x.get<float>(i);
                    dat->rec_z[i] = rec_z.get<float>(i);
                }
            }
            printf("rec %f %f %f %f\n",dat->src_x[0],dat->src_z[0],dat->rec_x[0],dat->rec_z[0]);
        }
        jsonBuffer.clear();
    }
    return dat;
}
float *importData(char *path, int *len){
    char fpath[50] = "externaltools/";
    strcat(fpath, path);
    *len = 0;
    float *data = 0;
    FILE *datafile = fopen(fpath,"r");
    if(datafile){
        while(!feof(datafile)){
            float datavalue;
            fscanf(datafile, "%f\n", &datavalue);
            *len = *len + 1;
        }
        fclose(datafile);

        datafile = fopen(fpath,"r");
        data = mat::createHost(*len);
        for(int i=0; i<*len; i++){
            fscanf(datafile, "%f\n", data + i);
        }
        fclose(datafile);
    }
    return data;
}
void exportData(float *data, int len, char *fname){
    char buffer[50] = "externaltools/";
    strcat(buffer, fname);
    FILE *file = fopen(buffer, "w");
    for(int i = 0; i < len; i++){
        fprintf(file, "%f\n", data[i]);
    }
    fclose(file);
}
void defineComputationalDomain(fdat *dat){
    dat->dx = dat->Lx / (dat->nx - 1);
    dat->dz = dat->Lz / (dat->nz - 1);
}
float *makeSourceTimeFunction(fdat *dat, int index){
    float *stf = mat::createHost(dat->nt);
    float max = 0;
    float alfa = 2 * dat->tauw_0[index] / dat->tauw[index];
    for(int i = 0; i < dat->nt; i++){
        float t = i * dat->dt;
        switch(dat -> stf_type[index]){
            case 2:{
                stf[i] = (-2 * pow(alfa, 3) / pi) * (t - dat->tee_0[index]) * exp(-pow(alfa, 2) * pow(t - dat->tee_0[index], 2));
                break;
            }
            // other stf: modify later
        }

        if(fabs(stf[i]) > max){
            max = fabs(stf[i]);
        }
    }
    if(max > 0){
        for(int i = 0; i < dat->nt; i++){
            stf[i] /= max;
        }
    }
    return stf;
}
void prepareSTF(fdat *dat){
    float *t = mat::createHost(dat->nt);
    int nt = dat->nt;
    for(int i = 0; i < nt; i++){
        t[i] = i * dat->dt;
    }

    dat->stf_x = mat::createHost(dat->nsrc, nt);
    dat->stf_y = mat::createHost(dat->nsrc, nt);
    dat->stf_z = mat::createHost(dat->nsrc, nt);
    float amp = dat->source_amplitude / dat->dx / dat->dz;
    for(int i=0; i < dat->nsrc; i++){
        float *stfn = makeSourceTimeFunction(dat, i);
        float px = dat->stf_PSV_x[i];
        float pz = dat->stf_PSV_z[i];
        float norm = sqrt(pow(px,2) + pow(pz,2));
        for(int j = 0; j < nt; j++){
            dat->stf_x[i][j] = amp * stfn[j] * px / norm;
            dat->stf_y[i][j] = amp * stfn[j];
            dat->stf_z[i][j] = amp * stfn[j] * pz / norm;
        }
    }
}
void checkArgs(fdat *dat){
    // int len;
    // add input file option: modify later
    // if update_params == 1  defineMaterialParameters here
    // float *stfall = importData("stf", &len);
    // if(len > 0){
    //     stf.stf_x = stfall;
    //     return stf;
    // }
    dat->update_params = 0;
    float *t = mat::createHost(dat->nt);
    for(int i = 0; i < dat->nt; i++){
        t[i] = i * dat->dt;
    }
    exportData(t, dat->nt, "t");
    prepareSTF(dat);
}
void defineMaterialParameters(fdat *dat){
    // more model_type: modify later
    int nx = dat->nx;
    int nz = dat->nz;
    switch(dat->model_type){
        case 1:{
            dat->rho = mat::createHost(nx, nz, 3000);
            dat->mu = mat::createHost(nx, nz, 4.8e10);
            dat->lambda = mat::createHost(nx, nz, 4.8e10);
            break;
        }
        case 10:{
            dat->rho = mat::createHost(nx, nz, 2600);
            dat->mu = mat::createHost(nx, nz, 2.66e10);
            dat->lambda = mat::createHost(nx, nz, 3.42e10);
            break;
        }
    }
}
void runWaveFieldPropagation(void){

}
void runForward(void){
    fdat *dat = importData();
    defineComputationalDomain(dat);
    checkArgs(dat);
    exportData(dat->stf_z[0],dat->nt,"stf_z"); // modify later
    if(!dat->update_params){
        defineMaterialParameters(dat);
    }
    //next: compute_indices
}

int main(int argc , char *argv[]){
    for(int i = 0; i< argc; i++){
        if(strcmp(argv[i],"runForward") == 0){
            runForward();
        }
    }

    return 0;
}
