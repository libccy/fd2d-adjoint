#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "ArduinoJson.h"

#define devij int i = blockIdx.x, j = threadIdx.x + blockIdx.y * blockDim.x

const float pi = 3.1415927;
const int nbt = 1;

typedef struct{
    int nx;
    int nz;
    int nt;
    float dx;
    float dz;
    float dt;
    float Lx;
    float Lz;

    int sfe;
    int nsfe;
    int order;
    int model_type;
    int wave_propagation_sh;
    int wave_propagation_psv;
    int simulation_mode;
    int use_given_model;
    int use_given_stf;
    float source_amplitude;

    int absorb_left;
    int absorb_right;
    int absorb_top;
    int absorb_bottom;
    float absorb_width;

    int isrc;
    int nsrc;
    int nrec;
    int obs_type;

    int *stf_type;  // host
    float *stf_PSV_x;  // host
    float *stf_PSV_z;  // host
    float *tauw_0;  // host
    float *tauw;  // host
    float *tee_0;  // host
    float *f_min;  // host
    float *f_max;  // host

    float *src_x;
    float *src_z;
    float *rec_x;
    float *rec_z;

    int *src_x_id;
    int *src_z_id;
    int *rec_x_id;
    int *rec_z_id;

    float **stf_x;
    float **stf_y;
    float **stf_z;
    float **adstf_x;
    float **adstf_y;
    float **adstf_z;

    float **lambda;
    float **mu;
    float **rho;
    float **absbound;

    float **ux;
    float **uy;
    float **uz;
    float **vx;
    float **vy;
    float **vz;

    float **sxx;
    float **sxy;
    float **sxz;
    float **szy;
    float **szz;

    float **dsx;
    float **dsy;
    float **dsz;
    float **dvxdx;
    float **dvxdz;
    float **dvydx;
    float **dvydz;
    float **dvzdx;
    float **dvzdz;

    float **dvxdx_fw;
    float **dvxdz_fw;
    float **dvydx_fw;
    float **dvydz_fw;
    float **dvzdx_fw;
    float **dvzdz_fw;

    float **K_lambda;
    float **K_mu;
    float **K_rho;

    float **v_rec_x;
    float **v_rec_y;
    float **v_rec_z;

    float ***ux_forward;  // host
    float ***uy_forward;  // host
    float ***uz_forward;  // host
    float ***vx_forward;  // host
    float ***vy_forward;  // host
    float ***vz_forward;  // host
} fdat;

namespace mat{
    __global__ void _setValue(float *mat, const float init){
        int i = blockIdx.x;
        mat[i] = init;
    }
    __global__ void _setValue(float **mat, const float init){
        devij;
        mat[i][j] = init;
    }
    __global__ void _setValue(float ***mat, const float init, const int p){
        devij;
        mat[p][i][j] = init;
    }
    __global__ void _setPointerValue(float **mat, float *data, const int n){
        int i = blockIdx.x;
        mat[i] = data + n * i;
    }
    __global__ void _setPointerValue(float ***mat, float **data, const int i){
        mat[i] = data;
    }


    float *init(float *mat, const int m, const float init){
        mat::_setValue<<<m, 1>>>(mat, init);
        return mat;
    }
    float **init(float **mat, const int m, const int n, const float init){
        dim3 dimGrid(m, nbt);
        mat::_setValue<<<dimGrid, n / nbt>>>(mat, init);
        return mat;
    }
    float ***init(float ***mat, const int p, const int m, const int n, const float init){
        dim3 dimGrid(m, nbt);
        for(int i = 0; i < p; i++){
            mat::_setValue<<<dimGrid, n / nbt>>>(mat, init, i);
        }
        return mat;
    }
    float *initHost(float *mat, const int m, const float init){
        for(int i = 0; i < m; i++){
            mat[i] = init;
        }
        return mat;
    }
    float **initHost(float **mat, const int m, const int n, const float init){
        for(int i = 0; i < m; i++){
            for(int j = 0; j < n; j++){
                mat[i][j] = init;
            }
        }
        return mat;
    }
    float ***initHost(float ***mat, const int p, const int m, const int n, float init){
        for(int k = 0; k < p; k++){
            for(int i = 0; i < m; i++){
                for(int j = 0; j < n; j++){
                    mat[k][i][j] = init;
                }
            }
        }
        return mat;
    }

    float *create(const int m) {
    	float *data;
    	hipMalloc((void **)&data, m * sizeof(float));
    	return data;
    }
    float **create(const int m, const int n){
    	float *data = mat::create(m * n);
        float **mat;
        hipMalloc((void **)&mat, m * sizeof(float *));
        mat::_setPointerValue<<<m, 1>>>(mat, data, n);
    	return mat;
    }
    float ***create(const int p, const int m, const int n){
        float ***mat;
        hipMalloc((void **)&mat, p * sizeof(float **));
        for(int i = 0; i < p; i++){
            mat::_setPointerValue<<<1,1>>>(mat, mat::create(m, n), i);
        }
        return mat;
    }
    float *createHost(const int m) {
    	return (float *)malloc(m * sizeof(float));
    }
    float **createHost(const int m, const int n){
        float *data = mat::createHost(m * n);
    	float **mat = (float **)malloc(m * sizeof(float *));
    	for(int i  =0; i < m; i++){
    		mat[i] = data + n * i;
    	}
    	return mat;
    }
    float ***createHost(const int p, const int m, const int n){
        float ***mat = (float ***)malloc(p * sizeof(float **));
        for(int i = 0; i < p; i++){
            mat[i] = mat::createHost(m, n);
        }
        return mat;
    }
    int *createInt(const int m){
        int *a;
    	hipMalloc((void**)&a, m * sizeof(int));
    	return a;
    }
    int *createIntHost(const int m) {
    	return (int *)malloc(m * sizeof(int));
    }

    void copyHostToDevice(float *d_a, const float *a, const int m){
        hipMemcpy(d_a, a , m * sizeof(float), hipMemcpyHostToDevice);
    }
    void copyHostToDevice(float **pd_a, float **pa, const int m, const int n){
        float **phd_a=(float **)malloc(sizeof(float *));
        hipMemcpy(phd_a, pd_a , sizeof(float *), hipMemcpyDeviceToHost);
        hipMemcpy(*phd_a, *pa , m * n * sizeof(float), hipMemcpyHostToDevice);
    }
    void copyHostToDevice(float ***pd_a, float ***pa, const int p, const int m, const int n){
        float ***phd_a=(float ***)malloc(p * sizeof(float **));
        hipMemcpy(phd_a, pd_a, p * sizeof(float **), hipMemcpyDeviceToHost);
        for(int i = 0; i < p; i++){
            mat::copyHostToDevice(phd_a[i], pa[i], m, n);
        }
    }
    void copyDeviceToHost(float *a, const float *d_a, const int m){
        hipMemcpy(a, d_a , m * sizeof(float), hipMemcpyDeviceToHost);
    }
    void copyDeviceToHost(float **pa, float **pd_a, const int m, const int n){
        float **phd_a=(float **)malloc(sizeof(float *));
        hipMemcpy(phd_a, pd_a , sizeof(float *), hipMemcpyDeviceToHost);
        hipMemcpy(*pa, *phd_a , m * n * sizeof(float), hipMemcpyDeviceToHost);
    }
    void copyDeviceToHost(float ***pa, float ***pd_a, const int p, const int m, const int n){
        float ***phd_a=(float ***)malloc(p * sizeof(float **));
        hipMemcpy(phd_a, pd_a, p * sizeof(float **), hipMemcpyDeviceToHost);
        for(int i = 0; i < p; i++){
            mat::copyDeviceToHost(pa[i], phd_a[i], m, n);
        }
    }

    void freeMat(float *mat){
        free(mat);
    }
    void freeMat(float **mat){
        free(*mat);
        free(mat);
    }

    void read(float *data, int n, char *fname){
        char buffer[50] = "externaltools/";
        strcat(buffer, fname);
        FILE *file = fopen(buffer, "rb");
        fwrite(data, sizeof(float), n, file);
        fclose(file);
    }
    void write(float *data, int n, char *fname){
        char buffer[50] = "externaltools/";
        strcat(buffer, fname);
        FILE *file = fopen(buffer, "wb");
        fwrite(data, sizeof(float), n, file);
        fclose(file);
    }
    void write(float **data, int m, int n, char *fname){
        char buffer[50] = "externaltools/";
        strcat(buffer, fname);
        FILE *file = fopen(buffer, "wb");
        for(int i = 0; i < m; i++){
            fwrite(data[i], sizeof(float), n, file);
        }
        fclose(file);
    }
    void write(float ***data, int p, int m, int n, char *fname){
        char buffer[50] = "externaltools/";
        strcat(buffer, fname);
        FILE *file = fopen(buffer, "wb");
        for(int k = 0; k < p; k++){
            for(int i = 0; i < m; i++){
                fwrite(data[k][i], sizeof(float), n, file);
            }
        }
        fclose(file);
    }
}

__global__ void divSY(float **dsy, float **sxy, float **szy, float dx, float dz, int nx, int nz){
    devij;
    if(i >= 2 && i < nx - 2){
        dsy[i][j] = 9*(sxy[i][j]-sxy[i-1][j])/(8*dx)-(sxy[i+1][j]-sxy[i-2][j])/(24*dx);
    }
    else{
        dsy[i][j] = 0;
    }
    if(j >= 2 && j < nz - 2){
        dsy[i][j] += 9*(szy[i][j]-szy[i][j-1])/(8*dz)-(szy[i][j+1]-szy[i][j-2])/(24*dz);
    }
}
__global__ void divSXZ(float **dsx, float **dsz, float **sxx, float **szz, float **sxz, float dx, float dz, int nx, int nz){
    devij;
    if(i >= 2 && i < nx - 2){
        dsx[i][j] = 9*(sxx[i][j]-sxx[i-1][j])/(8*dx)-(sxx[i+1][j]-sxx[i-2][j])/(24*dx);
        dsz[i][j] = 9*(sxz[i][j]-sxz[i-1][j])/(8*dx)-(sxz[i+1][j]-sxz[i-2][j])/(24*dx);
    }
    else{
        dsx[i][j] = 0;
        dsz[i][j] = 0;
    }
    if(j >= 2 && j < nz - 2){
        dsx[i][j] += 9*(sxz[i][j]-sxz[i][j-1])/(8*dz)-(sxz[i][j+1]-sxz[i][j-2])/(24*dz);
        dsz[i][j] += 9*(szz[i][j]-szz[i][j-1])/(8*dz)-(szz[i][j+1]-szz[i][j-2])/(24*dz);
    }
}
__global__ void divVY(float **dvydx, float **dvydz, float **vy, float dx, float dz, int nx, int nz){
    devij;
    if(i >= 1 && i < nx - 2){
        dvydx[i][j] = 9*(vy[i+1][j]-vy[i][j])/(8*dx)-(vy[i+2][j]-vy[i-1][j])/(24*dx);
    }
    else{
        dvydx[i][j] = 0;
    }
    if(j >= 1 && j < nz - 2){
        dvydz[i][j] = 9*(vy[i][j+1]-vy[i][j])/(8*dz)-(vy[i][j+2]-vy[i][j-1])/(24*dz);
    }
    else{
        dvydz[i][j] = 0;
    }
}
__global__ void divVXZ(float **dvxdx, float **dvxdz, float **dvzdx, float **dvzdz, float **vx, float **vz, float dx, float dz, int nx, int nz){
    devij;
    if(i >= 1 && i < nx - 2){
        dvxdx[i][j] = 9*(vx[i+1][j]-vx[i][j])/(8*dx)-(vx[i+2][j]-vx[i-1][j])/(24*dx);
        dvzdx[i][j] = 9*(vz[i+1][j]-vz[i][j])/(8*dx)-(vz[i+2][j]-vz[i-1][j])/(24*dx);
    }
    else{
        dvxdx[i][j] = 0;
        dvzdx[i][j] = 0;
    }
    if(j >= 1 && j < nz - 2){
        dvxdz[i][j] = 9*(vx[i][j+1]-vx[i][j])/(8*dz)-(vx[i][j+2]-vx[i][j-1])/(24*dz);
        dvzdz[i][j] = 9*(vz[i][j+1]-vz[i][j])/(8*dz)-(vz[i][j+2]-vz[i][j-1])/(24*dz);
    }
    else{
        dvxdz[i][j] = 0;
        dvzdz[i][j] = 0;
    }
}

__global__ void addSTF(float **dsx, float **dsy, float **dsz, float **stf_x, float **stf_y, float **stf_z,
    int *src_x_id, int *src_z_id, int isrc, int sh, int psv, int it){
    int is = blockIdx.x;
    int xs = src_x_id[is];
    int zs = src_z_id[is];
    if(isrc < 0 || isrc == is){
        if(sh){
            dsy[xs][zs] += stf_y[is][it];
        }
        if(psv){
            dsx[xs][zs] += stf_x[is][it];
            dsz[xs][zs] += stf_z[is][it];
        }
    }
}
__global__ void saveV(float **v_rec_x, float **v_rec_y, float **v_rec_z, float **vx, float **vy, float **vz,
    int *rec_x_id, int *rec_z_id, int sh, int psv, int it){
    int ir = blockIdx.x;
    int xr = rec_x_id[ir];
    int zr = rec_z_id[ir];
    if(sh){
        v_rec_y[ir][it] = vy[xr][zr];
    }
    if(psv){
        v_rec_x[ir][it] = vx[xr][zr];
        v_rec_z[ir][it] = vz[xr][zr];
    }
}
__global__ void updateV(float **v, float **ds, float **rho, float **absbound, float dt){
    devij;
    v[i][j] = absbound[i][j] * (v[i][j] + dt * ds[i][j] / rho[i][j]);
}
__global__ void updateSY(float **sxy, float **szy, float **dvydx, float **dvydz, float **mu, float dt){
    devij;
    sxy[i][j] += dt * mu[i][j] * dvydx[i][j];
    szy[i][j] += dt * mu[i][j] * dvydz[i][j];
}
__global__ void updateSXZ(float **sxx, float **szz, float **sxz, float **dvxdx, float **dvxdz, float **dvzdx, float **dvzdz,
    float **lambda, float **mu, float dt){
    devij;
    sxx[i][j] += dt * ((lambda[i][j] + 2 * mu[i][j]) * dvxdx[i][j] + lambda[i][j] * dvzdz[i][j]);
    szz[i][j] += dt * ((lambda[i][j] + 2 * mu[i][j]) * dvzdz[i][j] + lambda[i][j] * dvxdx[i][j]);
    sxz[i][j] += dt * (mu[i][j] * (dvxdz[i][j] + dvzdx[i][j]));
}
__global__ void updateU(float **u, float **v, float dt){
    devij;
    u[i][j] += v[i][j] * dt;
}
__global__ void interactionRhoY(float **K_rho, float **vy, float **vy_fw, float tsfe){
    devij;
    K_rho[i][j] -= vy_fw[i][j] * vy[i][j] * tsfe;
}
__global__ void interactionRhoXZ(float **K_rho, float **vx, float **vx_fw, float **vz, float **vz_fw, float tsfe){
    devij;
    K_rho[i][j] -= (vx_fw[i][j] * vx[i][j] + vz_fw[i][j] * vz[i][j]) * tsfe;
}
__global__ void interactionMuY(float **K_mu, float **dvydx, float **dvydx_fw, float **dvydz, float **dvydz_fw, float tsfe){
    devij;
    K_mu[i][j] -= (dvydx[i][j] * dvydx_fw[i][j] + dvydz[i][j] * dvydz_fw[i][j]) * tsfe;
}
__global__ void interactionMuXZ(float **K_mu, float **dvxdx, float **dvxdx_fw, float **dvxdz, float **dvxdz_fw,
    float **dvzdx, float **dvzdx_fw, float **dvzdz, float **dvzdz_fw, float tsfe){
    devij;
    K_mu[i][j] -= (2 * dvxdx[i][j] * dvxdx_fw[i][j] + 2 * dvzdz[i][j] * dvzdz_fw[i][j] +
        (dvxdz[i][j] + dvzdx[i][j]) * (dvzdx_fw[i][j] + dvxdz_fw[i][j])) * tsfe;
}
__global__ void interactionLambdaXZ(float **K_lambda, float **dvxdx, float **dvxdx_fw, float **dvzdz, float **dvzdz_fw, float tsfe){
    devij;
    K_lambda[i][j] -= ((dvxdx[i][j] + dvzdz[i][j]) * (dvxdx_fw[i][j] + dvzdz_fw[i][j])) * tsfe;
}

__global__ void computeIndices(int *coord_n_id, float *coord_n, float Ln, float n){
    int i = blockIdx.x;
    coord_n_id[i] = (int)(coord_n[i] / Ln * (n - 1) + 0.5);
}
__global__ void initialiseAbsorbingBoundaries(float **absbound, float width,
    int absorb_left, int absorb_right, int absorb_bottom, int absorb_top,
    float Lx, float Lz, float dx, float dz){
    devij;
    absbound[i][j] = 1;

    float X = i * dx;
    float Z = j * dz;
    if(absorb_left){
        if(X < width){
            absbound[i][j] *= exp(-pow((X - width) / (2 * width), 2));
        }
    }
    if(absorb_right){
        if(X > Lx - width){
            absbound[i][j] *= exp(-pow((X - (Lx - width)) / (2 * width), 2));
        }
    }
    if(absorb_bottom){
        if(Z < width){
            absbound[i][j] *= exp(-pow((Z - width) / (2 * width), 2));
        }
    }
    if(absorb_top){
        if(Z > Lz - width){
            absbound[i][j] *= exp(-pow((Z - (Lz - width)) / (2 * width), 2));
        }
    }
}
__global__ void prepareAdjointSTF(float **adstf, float **u_syn, float **u_obs, float *tw, int nt){
    int it = blockIdx.x;
    int irec = threadIdx.x;
    adstf[irec][nt - it - 1] = (u_syn[irec][it] - u_obs[irec][it]) * tw[it] * 2;
}
__global__ void normKernel(float **rho, float **mu, float **lambda, float misfit_init){
    devij;
    rho[i][j] /= misfit_init;
    mu[i][j] /= misfit_init;
    lambda[i][j] /= misfit_init;
}

fdat *importData(void){
    fdat *dat = new fdat;
    FILE *datfile = fopen("externaltools/config","r");

    char *buffer = 0;
    long length;

    fseek (datfile, 0, SEEK_END);
    length = ftell (datfile);
    fseek (datfile, 0, SEEK_SET);
    buffer = (char *)malloc (length + 1);
    fread (buffer, 1, length, datfile);
    buffer[length] = '\0';

    fclose(datfile);

    if (buffer){
        DynamicJsonBuffer jsonBuffer;
        JsonObject& root = jsonBuffer.parseObject(buffer);
        if (!root.success()){
            printf("parseObject() failed\n");
        }
        else{
            dat->nx = root["nx"];
            dat->nz = root["nz"];
            dat->nt = root["nt"];
            dat->dt = root["dt"];
            dat->Lx = root["Lx"];
            dat->Lz = root["Lz"];
            dat->sfe = root["sfe"];

            dat->model_type = root["model_type"];
            dat->use_given_model = root["use_given_model"];
            dat->use_given_stf = root["use_given_stf"];
            dat->source_amplitude = root["source_amplitude"];
            dat->order = root["order"]; // order = 2: later

            dat->absorb_left = root["absorb_left"];
            dat->absorb_right = root["absorb_right"];
            dat->absorb_top = root["absorb_top"];
            dat->absorb_bottom = root["absorb_bottom"];
            dat->absorb_width = root["width"];

            {
                const char* wave_propagation_type = root["wave_propagation_type"].as<char*>();
                if(strcmp(wave_propagation_type,"SH") == 0){
                    dat->wave_propagation_sh = 1;
                    dat->wave_propagation_psv = 0;
                }
                else if(strcmp(wave_propagation_type,"PSV") == 0){
                    dat->wave_propagation_sh = 0;
                    dat->wave_propagation_psv = 1;
                }
                else if(strcmp(wave_propagation_type,"both") == 0){
                    dat->wave_propagation_sh = 1;
                    dat->wave_propagation_psv = 1;
                }
                else{
                    dat->wave_propagation_sh = 0;
                    dat->wave_propagation_psv = 0;
                }
            }

            {
                int single_src = root["src_info"].is<JsonObject>();
                dat->nsrc = single_src?1:root["src_info"].size();

                float *src_x = mat::createHost(dat->nsrc);
                float *src_z = mat::createHost(dat->nsrc);

                dat->stf_type = mat::createIntHost(dat->nsrc);
                dat->stf_PSV_x = mat::createHost(dat->nsrc);
                dat->stf_PSV_z = mat::createHost(dat->nsrc);
                dat->tauw_0 = mat::createHost(dat->nsrc);
                dat->tauw = mat::createHost(dat->nsrc);
                dat->tee_0 = mat::createHost(dat->nsrc);
                dat->f_min = mat::createHost(dat->nsrc);
                dat->f_max = mat::createHost(dat->nsrc);

                for(int isrc = 0; isrc < dat->nsrc; isrc++){
                    JsonObject& src = single_src?root["src_info"]:((JsonArray&)root["src_info"]).get<JsonObject>(isrc);
                    src_x[isrc] = src["loc_x"];
                    src_z[isrc] = src["loc_z"];

                    dat->stf_PSV_x[isrc] = src["stf_PSV"][0];
                    dat->stf_PSV_z[isrc] = src["stf_PSV"][1];
                    dat->tauw_0[isrc] = src["tauw_0"];
                    dat->tauw[isrc] = src["tauw"];
                    dat->tee_0[isrc] = src["tee_0"];
                    dat->f_min[isrc] = src["f_min"];
                    dat->f_max[isrc] = src["f_max"];

                    const char* stf_type_str = src["stf_type"].as<char*>();
                    if(strcmp(stf_type_str,"delta") == 0){
                        dat->stf_type[isrc] = 0;
                    }
                    else if(strcmp(stf_type_str,"delta_bp") == 0){
                        dat->stf_type[isrc] = 1;
                    }
                    else if(strcmp(stf_type_str,"ricker") == 0){
                        dat->stf_type[isrc] = 2;
                    }
                    else if(strcmp(stf_type_str,"heaviside_bp") == 0){
                        dat->stf_type[isrc] = 3;
                    }
                    else{
                        dat->stf_type[isrc] = -1;
                    }
                }

                dat->src_x = mat::create(dat->nsrc);
                dat->src_z = mat::create(dat->nsrc);

                mat::copyHostToDevice(dat->src_x, src_x, dat->nsrc);
                mat::copyHostToDevice(dat->src_z, src_z, dat->nsrc);

                mat::freeMat(src_x);
                mat::freeMat(src_z);
            }

            {
                int single_rec = root["rec_x"].is<float>();
                dat->nrec = single_rec?1:root["rec_x"].size();

                float *rec_x = mat::createHost(dat->nrec);
                float *rec_z = mat::createHost(dat->nrec);

                for(int irec = 0; irec < dat->nrec; irec++){
                    rec_x[irec] = single_rec?root["rec_x"]:((JsonArray&)root["rec_x"]).get<float>(irec);
                    rec_z[irec] = single_rec?root["rec_z"]:((JsonArray&)root["rec_z"]).get<float>(irec);
                }

                dat->rec_x = mat::create(dat->nrec);
                dat->rec_z = mat::create(dat->nrec);

                mat::copyHostToDevice(dat->rec_x, rec_x, dat->nrec);
                mat::copyHostToDevice(dat->rec_z, rec_z, dat->nrec);

                mat::freeMat(rec_x);
                mat::freeMat(rec_z);
            }
        }
        jsonBuffer.clear();
    }
    return dat;
}
void checkMemoryUsage(){
    size_t free_byte ;
    size_t total_byte ;
    hipMemGetInfo( &free_byte, &total_byte ) ;
    float free_db = (float)free_byte ;
    float total_db = (float)total_byte ;
    float used_db = total_db - free_db ;

    printf("memory usage: %.1fMB / %.1fMB\n", used_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}
void makeSourceTimeFunction(fdat *dat, float *stf, int index){
    float max = 0;
    float alfa = 2 * dat->tauw_0[index] / dat->tauw[index];
    for(int it = 0; it < dat->nt; it++){
        float t = it * dat->dt;
        switch(dat -> stf_type[index]){
            case 2:{
                stf[it] = (-2 * pow(alfa, 3) / pi) * (t - dat->tee_0[index]) * exp(-pow(alfa, 2) * pow(t - dat->tee_0[index], 2));
                break;
            }
            // other stf: later
        }

        if(fabs(stf[it]) > max){
            max = fabs(stf[it]);
        }
    }
    if(max > 0){
        for(int it = 0; it < dat->nt; it++){
            stf[it] /= max;
        }
    }
}
void prepareSTF(fdat *dat){
    int &nt = dat->nt;
    float amp = dat->source_amplitude / dat->dx / dat->dz;
    float **stf_x = mat::createHost(dat->nsrc, dat->nt);
    float **stf_y = mat::createHost(dat->nsrc, dat->nt);
    float **stf_z = mat::createHost(dat->nsrc, dat->nt);
    float *stfn = mat::createHost(dat->nt);

    for(int isrc = 0; isrc < dat->nsrc; isrc++){
        makeSourceTimeFunction(dat, stfn, isrc);
        float px = dat->stf_PSV_x[isrc];
        float pz = dat->stf_PSV_z[isrc];
        float norm = sqrt(pow(px,2) + pow(pz,2));
        for(int it = 0; it < nt; it++){
            stf_x[isrc][it] = amp * stfn[it] * px / norm;
            stf_y[isrc][it] = amp * stfn[it];
            stf_z[isrc][it] = amp * stfn[it] * pz / norm;
        }
    }

    mat::copyHostToDevice(dat->stf_x, stf_x, dat->nsrc, dat->nt);
    mat::copyHostToDevice(dat->stf_y, stf_y, dat->nsrc, dat->nt);
    mat::copyHostToDevice(dat->stf_z, stf_z, dat->nsrc, dat->nt);

    mat::freeMat(stf_x);
    mat::freeMat(stf_y);
    mat::freeMat(stf_z);
    mat::freeMat(stfn);
}
void defineMaterialParameters(fdat *dat){
    // other model_type: later
    int &nx = dat->nx;
    int &nz = dat->nz;
    switch(dat->model_type){
        case 1:{
            mat::init(dat->rho, nx, nz, 3000);
            mat::init(dat->mu, nx, nz, 4.8e10);
            mat::init(dat->lambda, nx, nz, 4.8e10);
            break;
        }
        case 10:{
            mat::init(dat->rho, nx, nz, 2600);
            mat::init(dat->mu, nx, nz, 2.66e10);
            mat::init(dat->lambda, nx, nz, 3.42e10);
            break;
        }
        case 13:{
            mat::init(dat->mu, nx, nz, 2.66e10);
            mat::init(dat->lambda, nx, nz, 3.42e10);

            float rho = 2600;
            float mu = 2.66e10;
            float lambda = 3.42e10;
            float vp = sqrt((lambda + 2*mu) / rho);
            float vs = sqrt(mu / rho);
            int left = (int)((float)nx / 2 - (float)nx / 20 + 0.5);
            int right = (int)((float)nx / 2 + (float)nx / 20 + 0.5);
            int bottom = (int)((float)nz / 2 - (float)nz / 20 + 0.5);
            int top = (int)((float)nz / 2 + (float)nz / 20 + 0.5);

            float **rho2 = mat::createHost(nx, nz);
            mat::initHost(rho2, nx, nz, 2600);
            for(int i = left; i < right; i++){
                for(int j = bottom; j < top; j++){
                    rho2[i][j] = 3600;
                }
            }
            mat::copyHostToDevice(dat->rho, rho2, nx, nz);
            mat::freeMat(rho2);
        }
    }
}
void initialiseDynamicFields(fdat *dat){
    int &nx = dat->nx;
    int &nz = dat->nz;
    if(dat->wave_propagation_sh){
        mat::init(dat->vy, nx, nz, 0);
        mat::init(dat->uy, nx, nz, 0);
        mat::init(dat->sxy, nx, nz, 0);
        mat::init(dat->szy, nx, nz, 0);
    }
    if(dat->wave_propagation_psv){
        mat::init(dat->vx, nx, nz, 0);
        mat::init(dat->vz, nx, nz, 0);
        mat::init(dat->ux, nx, nz, 0);
        mat::init(dat->uz, nx, nz, 0);
        mat::init(dat->sxx, nx, nz, 0);
        mat::init(dat->szz, nx, nz, 0);
        mat::init(dat->sxz, nx, nz, 0);
    }
}
void initialiseKernels(fdat *dat){
    int &nx = dat->nx;
    int &nz = dat->nz;
    mat::init(dat->K_lambda, nx, nz, 0);
    mat::init(dat->K_mu, nx, nz, 0);
    mat::init(dat->K_rho, nx, nz, 0);
}
void runWaveFieldPropagation(fdat *dat){
    int &sh = dat->wave_propagation_sh;
    int &psv = dat->wave_propagation_psv;
    int &mode = dat->simulation_mode;

    int &nx = dat->nx;
    int &nz = dat->nz;
    float &dx = dat->dx;
    float &dz = dat->dz;
    float &dt = dat->dt;

    dim3 dimGrid(nx, nbt);
    dim3 dimBlock(nz / nbt);

    initialiseDynamicFields(dat);

    for(int it = 0; it < dat->nt; it++){
        if(mode == 0){
            if((it + 1) % dat->sfe == 0){
                int isfe = dat->nsfe - (it + 1) / dat->sfe;
                if(sh){
                    mat::copyDeviceToHost(dat->uy_forward[isfe], dat->uy, nx, nz);
                }
                if(psv){
                    mat::copyDeviceToHost(dat->ux_forward[isfe], dat->ux, nx, nz);
                    mat::copyDeviceToHost(dat->uz_forward[isfe], dat->uz, nx, nz);
                }
            }
        }

        if(sh){
            divSY<<<dimGrid, dimBlock>>>(dat->dsy, dat->sxy, dat->szy, dx, dz, nx, nz);
        }
        if(psv){
            divSXZ<<<dimGrid, dimBlock>>>(dat->dsx, dat->dsz, dat->sxx, dat->szz, dat->sxz, dx, dz, nx, nz);
        }
        if(mode == 0){
            addSTF<<<dat->nsrc, 1>>>(
                dat->dsx, dat->dsy, dat->dsz, dat->stf_x, dat->stf_y, dat->stf_z,
                dat->src_x_id, dat->src_z_id, dat->isrc, sh, psv, it
            );
        }
        else if(mode == 1){
            addSTF<<<dat->nrec, 1>>>(
                dat->dsx, dat->dsy, dat->dsz, dat->adstf_x, dat->adstf_y, dat->adstf_z,
                dat->rec_x_id, dat->rec_z_id, -1, sh, psv, it
            );
        }
        if(sh){
            updateV<<<dimGrid, dimBlock>>>(dat->vy, dat->dsy, dat->rho, dat->absbound, dt);
            divVY<<<dimGrid, dimBlock>>>(dat->dvydx, dat->dvydz, dat->vy, dx, dz, nx, nz);
            updateSY<<<dimGrid, dimBlock>>>(dat->sxy, dat->szy, dat->dvydx, dat->dvydz, dat->mu, dt);
            updateU<<<dimGrid, dimBlock>>>(dat->uy, dat->vy, dt);
        }
        if(psv){
            updateV<<<dimGrid, dimBlock>>>(dat->vx, dat->dsx, dat->rho, dat->absbound, dt);
            updateV<<<dimGrid, dimBlock>>>(dat->vz, dat->dsz, dat->rho, dat->absbound, dt);
            divVXZ<<<dimGrid, dimBlock>>>(dat->dvxdx, dat->dvxdz, dat->dvzdx, dat->dvzdz, dat->vx, dat->vz, dx, dz, nx, nz);
            updateSXZ<<<dimGrid, dimBlock>>>(dat->sxx, dat->szz, dat->sxz, dat->dvxdx, dat->dvxdz, dat->dvzdx, dat->dvzdz, dat->lambda, dat->mu, dt);
            updateU<<<dimGrid, dimBlock>>>(dat->ux, dat->vx, dt);
            updateU<<<dimGrid, dimBlock>>>(dat->uz, dat->vz, dt);
        }
        if(mode == 0){
            if(dat->obs_type == 0){
                saveV<<<dat->nrec, 1>>>(
                    dat->v_rec_x, dat->v_rec_y, dat->v_rec_z, dat->vx, dat->vy, dat->vz,
                    dat->rec_x_id, dat->rec_z_id, sh, psv, it
                );
            }
            else if(dat->obs_type == 1){
                saveV<<<dat->nrec, 1>>>(
                    dat->v_rec_x, dat->v_rec_y, dat->v_rec_z, dat->ux, dat->uy, dat->uz,
                    dat->rec_x_id, dat->rec_z_id, sh, psv, it
                );
            }
            if((it + 1) % dat->sfe == 0){
                int isfe = dat->nsfe - (it + 1) / dat->sfe;
                if(sh){
                    mat::copyDeviceToHost(dat->vy_forward[isfe], dat->vy, nx, nz);
                }
                if(psv){
                    mat::copyDeviceToHost(dat->vx_forward[isfe], dat->vx, nx, nz);
                    mat::copyDeviceToHost(dat->vz_forward[isfe], dat->vz, nx, nz);
                }
            }
        }
        else if(mode == 1){
            if((it + dat->sfe) % dat->sfe == 0){
                // dsi -> ui_fw -> vi_fw
                int isfe = (it + dat->sfe) / dat->sfe - 1;
                float tsfe = dat->sfe * dt;
                if(sh){
                    mat::copyHostToDevice(dat->dsy, dat->uy_forward[isfe], nx, nz);
                    divVY<<<dimGrid, dimBlock>>>(dat->dvydx, dat->dvydz, dat->uy, dx, dz, nx, nz);
                    divVY<<<dimGrid, dimBlock>>>(dat->dvydx_fw, dat->dvydz_fw, dat->dsy, dx, dz, nx, nz);
                    mat::copyHostToDevice(dat->dsy, dat->vy_forward[isfe], nx, nz);
                    interactionRhoY<<<dimGrid, dimBlock>>>(dat->K_rho, dat->vy, dat->dsy, tsfe);
                    interactionMuY<<<dimGrid, dimBlock>>>(dat->K_mu, dat->dvydx, dat->dvydx_fw, dat->dvydz, dat->dvydz_fw, tsfe);
                }
                if(psv){
                    mat::copyHostToDevice(dat->dsx, dat->ux_forward[isfe], nx, nz);
                    mat::copyHostToDevice(dat->dsz, dat->uz_forward[isfe], nx, nz);
                    divVXZ<<<dimGrid, dimBlock>>>(
                        dat->dvxdx, dat->dvxdz, dat->dvzdx, dat->dvzdz,
                        dat->ux, dat->uz, dx, dz, nx, nz
                    );
                    divVXZ<<<dimGrid, dimBlock>>>(
                        dat->dvxdx_fw, dat->dvxdz_fw, dat->dvzdx_fw, dat->dvzdz_fw,
                        dat->dsx, dat->dsz, dx, dz, nx, nz
                    );

                    mat::copyHostToDevice(dat->dsx, dat->vx_forward[isfe], nx, nz);
                    mat::copyHostToDevice(dat->dsz, dat->vz_forward[isfe], nx, nz);
                    interactionRhoXZ<<<dimGrid, dimBlock>>>(dat->K_rho, dat->vx, dat->dsx, dat->vz, dat->dsz, tsfe);
                    interactionMuXZ<<<dimGrid, dimBlock>>>(
                        dat->K_mu, dat->dvxdx, dat->dvxdx_fw, dat->dvxdz, dat->dvxdz_fw,
                        dat->dvzdx, dat->dvzdx_fw, dat->dvzdz, dat->dvzdz_fw, tsfe
                    );
                    interactionLambdaXZ<<<dimGrid, dimBlock>>>(dat->K_lambda, dat->dvxdx, dat->dvxdx_fw, dat->dvzdz, dat->dvzdz_fw, tsfe);
                }
            }
        }
    }
}
void checkArgs(fdat *dat, int adjoint){
    int &sh = dat->wave_propagation_sh;
    int &psv = dat->wave_propagation_psv;

    int &nx = dat->nx;
    int &nz = dat->nz;

    if(dat->nt % dat->sfe != 0){
        dat->nt = dat->sfe * (int)((float)dat->nt / dat->sfe + 0.5);
    }
    dat->nsfe = dat->nt / dat->sfe;
    dat->dx = dat->Lx / (nx - 1);
    dat->dz = dat->Lz / (nz - 1);
    dat->obs_type = 0;

    if(sh){
        dat->vy = mat::create(nx, nz);
        dat->uy = mat::create(nx, nz);
        dat->sxy = mat::create(nx, nz);
        dat->szy = mat::create(nx, nz);
        dat->dsy = mat::create(nx, nz);
        dat->dvydx = mat::create(nx, nz);
        dat->dvydz = mat::create(nx, nz);

        dat->v_rec_y = mat::create(dat->nrec, dat->nt);
        dat->uy_forward = mat::createHost(dat->nsfe, nx, nz);
        dat->vy_forward = mat::createHost(dat->nsfe, nx, nz);
    }
    if(psv){
        dat->vx = mat::create(nx, nz);
        dat->vz = mat::create(nx, nz);
        dat->ux = mat::create(nx, nz);
        dat->uz = mat::create(nx, nz);
        dat->sxx = mat::create(nx, nz);
        dat->szz = mat::create(nx, nz);
        dat->sxz = mat::create(nx, nz);
        dat->dsx = mat::create(nx, nz);
        dat->dsz = mat::create(nx, nz);
        dat->dvxdx = mat::create(nx, nz);
        dat->dvxdz = mat::create(nx, nz);
        dat->dvzdx = mat::create(nx, nz);
        dat->dvzdz = mat::create(nx, nz);

        dat->v_rec_x = mat::create(dat->nrec, dat->nt);
        dat->v_rec_z = mat::create(dat->nrec, dat->nt);
        dat->ux_forward = mat::createHost(dat->nsfe, nx, nz);
        dat->uz_forward = mat::createHost(dat->nsfe, nx, nz);
        dat->vx_forward = mat::createHost(dat->nsfe, nx, nz);
        dat->vz_forward = mat::createHost(dat->nsfe, nx, nz);
    }

    dat->absbound = mat::create(nx, nz);
    dat->lambda = mat::create(nx, nz);
    dat->rho = mat::create(nx, nz);
    dat->mu = mat::create(nx, nz);

    dat->stf_x = mat::create(dat->nsrc, dat->nt);
    dat->stf_y = mat::create(dat->nsrc, dat->nt);
    dat->stf_z = mat::create(dat->nsrc, dat->nt);

    if(adjoint){
        if(sh){
            dat->dvydx_fw = mat::create(nx, nz);
            dat->dvydz_fw = mat::create(nx, nz);
        }
        if(psv){
            dat->dvxdx_fw = mat::create(nx, nz);
            dat->dvxdz_fw = mat::create(nx, nz);
            dat->dvzdx_fw = mat::create(nx, nz);
            dat->dvzdz_fw = mat::create(nx, nz);
        }

        dat->K_lambda = mat::create(nx, nz);
        dat->K_mu = mat::create(nx, nz);
        dat->K_rho = mat::create(nx, nz);

        dat->adstf_x = mat::create(dat->nrec, dat->nt);
        dat->adstf_y = mat::create(dat->nrec, dat->nt);
        dat->adstf_z = mat::create(dat->nrec, dat->nt);
    }

    dat->src_x_id = mat::createInt(dat->nsrc);
    dat->src_z_id = mat::createInt(dat->nsrc);
    dat->rec_x_id = mat::createInt(dat->nrec);
    dat->rec_z_id = mat::createInt(dat->nrec);

    computeIndices<<<dat->nsrc, 1>>>(dat->src_x_id, dat->src_x, dat->Lx, dat->nx);
    computeIndices<<<dat->nsrc, 1>>>(dat->src_z_id, dat->src_z, dat->Lz, dat->nz);
    computeIndices<<<dat->nrec, 1>>>(dat->rec_x_id, dat->rec_x, dat->Lx, dat->nx);
    computeIndices<<<dat->nrec, 1>>>(dat->rec_z_id, dat->rec_z, dat->Lz, dat->nz);

    dim3 dimGrid(nx, nbt);
    dim3 dimBlock(nz / nbt);
    initialiseAbsorbingBoundaries<<<dimGrid, dimBlock>>>(
        dat->absbound, dat->absorb_width,
        dat->absorb_left, dat->absorb_right, dat->absorb_bottom, dat->absorb_top,
        dat->Lx, dat->Lz, dat->dx, dat->dz
    );

    float *t = mat::createHost(dat->nt);
    for(int it = 0; it < dat->nt; it++){
        t[it] = it * dat->dt;
    }
    mat::write(t, dat->nt, "t");
}
void runForward(fdat *dat, int isrc){
    dat->simulation_mode = 0;
    dat->isrc = isrc;
    runWaveFieldPropagation(dat);

    // float **v_rec_x=mat::createHost(dat->nrec, dat->nt);
    // float **v_rec_z=mat::createHost(dat->nrec, dat->nt);
    // mat::copyDeviceToHost(v_rec_x, dat->v_rec_x, dat->nrec, dat->nt);
    // mat::copyDeviceToHost(v_rec_z, dat->v_rec_z, dat->nrec, dat->nt);
    // mat::write(v_rec_x, dat->nrec, dat->nt, "vx_rec");
    // mat::write(v_rec_z, dat->nrec, dat->nt, "vz_rec");
    // mat::write(dat->vx_forward, dat->nsfe, dat->nx, dat->nz, "vx");
    // mat::write(dat->vz_forward, dat->nsfe, dat->nx, dat->nz, "vz");
}
void runAdjoint(fdat *dat, int init_kernel){
    dat->simulation_mode = 1;
    if(init_kernel){
        initialiseKernels(dat);
    }
    runWaveFieldPropagation(dat);

    // float **rho = mat::createHost(dat->nx, dat->nz);
    // float **mu = mat::createHost(dat->nx, dat->nz);
    // float **lambda = mat::createHost(dat->nx, dat->nz);
    // mat::copyDeviceToHost(rho, dat->K_rho, dat->nx, dat->nz);
    // mat::copyDeviceToHost(mu, dat->K_mu, dat->nx, dat->nz);
    // mat::copyDeviceToHost(lambda, dat->K_lambda, dat->nx, dat->nz);
    // mat::write(rho, dat->nx, dat->nz, "rho");
    // mat::write(mu, dat->nx, dat->nz, "mu");
    // mat::write(lambda, dat->nx, dat->nz, "lambda");
    // mat::write(dat->vx_forward, dat->nsfe, dat->nx, dat->nz, "vx");
    // mat::write(dat->vz_forward, dat->nsfe, dat->nx, dat->nz, "vz");
}
float calculateMisfit(float *u_syn, float *u_obs, float *tw, float dt, int nt){
    float misfit = 0;
    for(int it = 1; it < nt; it++){
        float wavedif = (u_syn[it] - u_obs[it]) * tw[it];
        misfit += wavedif * wavedif * dt;
    }
    return misfit;
}
float *getTaperWeights(float dt, int nt){
    float t_end = (nt - 1) * dt;
    float taper_width = t_end / 10;
    float t_min = taper_width;
    float t_max = t_end - taper_width;

    float *tw = mat::createHost(nt);
    for(int it = 0; it < nt; it++){
        float t = it * dt;
        if(t <= t_min){
            tw[it] = 0.5 + 0.5 * cos(pi * (t_min - t) / (taper_width));
        }
        else if(t >= t_max){
            tw[it] = 0.5 + 0.5 * cos(pi * (t_max - t) / (taper_width));
        }
        else{
            tw[it] = 1;
        }
    }
    return tw;
}
void convertV2U(float ***v, int nsrc, int nrec, int nt, float dt){
    for(int isrc = 0; isrc < nsrc; isrc++){
        for(int irec = 0; irec < nrec; irec++){
            for(int it = 0; it < nt; it++){
                v[isrc][irec][it] *= dt;
                if(it > 0){
                    v[isrc][irec][it] += + v[isrc][irec][it-1];
                }
            }
        }
    }
}
void inversionRoutine(fdat *dat, float ***u_obs_x, float ***u_obs_z){
    int niter = 1; // move to dat: later

    int &nsrc = dat->nsrc;
    int &nrec = dat->nrec;
    // int &sh = dat->wave_propagation_sh; // sh: later
    // int &psv = dat->wave_propagation_psv;

    int &nx = dat->nx;
    int &nz = dat->nz;
    int &nt = dat->nt;
    float &dt = dat->dt;

    dim3 dimGrid(nx, nbt);
    dim3 dimBlock(nz / nbt);

    // prepare syn and obs
    dat->obs_type = 1;
    convertV2U(u_obs_x, nsrc, nrec, nt, dt);
    convertV2U(u_obs_z, nsrc, nrec, nt, dt);

    float **u_syn_x = mat::createHost(nrec, nt);
    float **u_syn_z = mat::createHost(nrec, nt);

    float **d_u_obs_x = mat::create(nrec, nt);
    float **d_u_obs_z = mat::create(nrec, nt);

    // taper weights
    float *tw = getTaperWeights(dt, nt);
    float *d_tw = mat::create(nt);
    mat::copyHostToDevice(d_tw, tw, nt);

    // kernels
    initialiseKernels(dat);

    float misfit_init;

    for(int iter = 0; iter < niter; iter++){
        float misfit = 0;
        for(int isrc = 0; isrc < nsrc; isrc++){
            runForward(dat, isrc);
            mat::copyDeviceToHost(u_syn_x, dat->v_rec_x, nrec, nt);
            mat::copyDeviceToHost(u_syn_z, dat->v_rec_z, nrec, nt);
            for(int irec = 0; irec < nrec; irec++){
                misfit += calculateMisfit(u_syn_x[irec], u_obs_x[isrc][irec], tw, dt, nt);
                misfit += calculateMisfit(u_syn_z[irec], u_obs_z[isrc][irec], tw, dt, nt);
            }

            // if(iter < niter - 1){
                mat::copyHostToDevice(d_u_obs_x, u_obs_x[isrc], nrec, nt);
                mat::copyHostToDevice(d_u_obs_z, u_obs_z[isrc], nrec, nt);
                prepareAdjointSTF<<<nt, dat->nrec>>>(dat->adstf_x, dat->v_rec_x, d_u_obs_x, d_tw, nt);
                prepareAdjointSTF<<<nt, dat->nrec>>>(dat->adstf_z, dat->v_rec_z, d_u_obs_z, d_tw, nt);
                mat::init(dat->adstf_y, nrec, nt, 0);
                runAdjoint(dat, 0);
            // }
        }
        printf("iter=%d misfit=%e\n", iter, misfit);
        if(iter == 0){
            misfit_init = misfit;
            misfit = 1;
        }
        else{
            misfit /= misfit_init;
        }

        // if(iter < niter - 1){
            normKernel<<<dimGrid, dimBlock>>>(dat->K_rho, dat->K_mu, dat->K_lambda, misfit_init);
            float **lambda = mat::createHost(nx,nz);
            float **mu = mat::createHost(nx,nz);
            float **rho = mat::createHost(nx,nz);
            mat::copyDeviceToHost(rho, dat->K_rho, dat->nx, dat->nz);
            mat::copyDeviceToHost(mu, dat->K_mu, dat->nx, dat->nz);
            mat::copyDeviceToHost(lambda, dat->K_lambda, dat->nx, dat->nz);
            mat::write(rho, dat->nx, dat->nz, "rho");
            mat::write(mu, dat->nx, dat->nz, "mu");
            mat::write(lambda, dat->nx, dat->nz, "lambda");
        // }
    }
}
void runSyntheticInvertion(fdat *dat){
    int &nsrc = dat->nsrc;
    int &nrec = dat->nrec;
    int &nt = dat->nt;

    checkArgs(dat, 1); // obs_type set to 0
    dat->model_type = 13; // true model: later
    prepareSTF(dat); //dat->use_given_stf, sObsPerFreq: later
    defineMaterialParameters(dat); //dat->use_given_model: later
    float ***v_obs_x=mat::createHost(nsrc, nrec, nt);
    float ***v_obs_z=mat::createHost(nsrc, nrec, nt);
    for(int isrc = 0; isrc < nsrc; isrc++){
        runForward(dat, isrc);
        mat::copyDeviceToHost(v_obs_x[isrc], dat->v_rec_x, nrec, nt);
        mat::copyDeviceToHost(v_obs_z[isrc], dat->v_rec_z, nrec, nt);
    }

    dat->model_type = 10;
    defineMaterialParameters(dat);
    inversionRoutine(dat, v_obs_x, v_obs_z);
}

int main(int argc , char *argv[]){
    fdat *dat = importData();
    if(argc == 1){
        runSyntheticInvertion(dat);
    }
    else{
        for(int i = 1; i< argc; i++){
            if(strcmp(argv[i],"run_forward") == 0){
                checkArgs(dat, 0);
                prepareSTF(dat);
                defineMaterialParameters(dat);
                runForward(dat, -1);
            }
        }
    }
    checkMemoryUsage();

    return 0;
}
