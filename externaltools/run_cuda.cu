#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "ArduinoJson.h"

#define devij(dimx, dimy) \
int i = blockIdx.x % dimx; \
int j = threadIdx.x + (blockIdx.x - i) / dimx * dimy / d_nbt;

const float pi = 3.1415927;
const int nbt = 8;
__constant__ int d_nbt = 8;

typedef struct{
    int nx;
    int nz;
    int nt;
    float dx;
    float dz;
    float dt;
    float Lx;
    float Lz;

    int sfe;
    int update_params;
    int model_type;
    float source_amplitude;

    int nsrc;
    int *stf_type;
    float *stf_PSV_x;
    float *stf_PSV_z;
    float *src_x;
    float *src_z;
    float *tauw_0;
    float *tauw;
    float *tee_0;
    float *f_min;
    float *f_max;

    float **stf_x;
    float **stf_y;
    float **stf_z;

    float **lambda;
    float **mu;
    float **rho;
} fdat;

namespace mat{
    __global__ void _setValue(float **mat, const float init, const int m, const int n){
        devij(m, n);
        mat[i][j] = init;
    }
    __global__ void _setValue(float *mat, const float init, const int m){
        int i = threadIdx.x;
        mat[i] = init;
    }
    __global__ void _setPointerValue(float **mat, float *data, const int n){
        int i = threadIdx.x;
        mat[i] = data + n * i;
    }
    __global__ void _setIntPointerValue(int **mat, int *data, const int n){
        int i = threadIdx.x;
        mat[i] = data + n * i;
    }

    float *create(const int m) {
    	float *data;
    	hipMalloc((void**)&data, m * sizeof(float));
    	return data;
    }
    float **create(const int m, const int n){
    	float *data;
    	hipMalloc((void**)&data, m * n * sizeof(float));
        float **mat;
        hipMalloc((void**)&mat, m * sizeof(float *));
        mat::_setPointerValue<<<1, m>>>(mat, data, n);
    	return mat;
    }
    float *createHost(const int m) {
    	return (float *)malloc(m * sizeof(float));
    }
    float **createHost(const int m, const int n){
    	float **mat = (float **)malloc(m * sizeof(float *));
    	float *data = (float *)malloc(m * n * sizeof(float));
    	for(int i  =0; i < m; i++){
    		mat[i] = data + n * i;
    	}
    	return mat;
    }
    int *createInt(const int m){
        int *a;
    	hipMalloc((void**)&a, m * sizeof(int));
    	return a;
    }
    int **createInt(const int m, const int n){
    	int *data;
    	hipMalloc((void**)&data, m * n * sizeof(int));
        int **mat;
        hipMalloc((void**)&mat, m * sizeof(int *));
        mat::_setIntPointerValue<<<1, m>>>(mat, data, n);
    	return mat;
    }
    int *createIntHost(const int m) {
    	return (int *)malloc(m * sizeof(int));
    }
    int **createIntHost(const int m, const int n){
    	int **mat = (int **)malloc(m * sizeof(int *));
    	int *data = (int *)malloc(m * n * sizeof(int));
    	for(int i  =0; i < m; i++){
    		mat[i] = data + n * i;
    	}
    	return mat;
    }

    float *init(float *mat, const float init, const int m){
        mat::_setValue<<<1, m>>>(mat, init, m);
        return mat;
    }
    float **init(float **mat, const float init, const int m, const int n){
        mat::_setValue<<<m * nbt, n / nbt>>>(mat, init, m, n);
        return mat;
    }
    float *initHost(float *mat, const float init, const int m){
        for(int i = 0; i < m; i++){
            mat[i] = init;
        }
        return mat;
    }
    float **initHost(float **mat, const float init, const int m, const int n){
        for(int i = 0; i < m; i++){
            for(int j = 0; j < n; j++){
                mat[i][j] = init;
            }
        }
        return mat;
    }

    void copyHostToDevice(float *d_a, const float *a, const int m){
        hipMemcpy(d_a, a , m * sizeof(float), hipMemcpyHostToDevice);
    }
    void copyHostToDevice(float **pd_a, float **pa, const int m, const int n){
        float **phd_a=(float **)malloc(sizeof(float *));
        hipMemcpy(phd_a, pd_a , sizeof(float *), hipMemcpyDeviceToHost);
        hipMemcpy(*phd_a, *pa , m * n * sizeof(float), hipMemcpyHostToDevice);
    }
    void copyDeviceToHost(float *a, const float *d_a, const int m){
        hipMemcpy(a, d_a , m * sizeof(float), hipMemcpyDeviceToHost);
    }
    void copyDeviceToHost(float **pa, float **pd_a, const int m, const int n){
        float **phd_a=(float **)malloc(sizeof(float *));
        hipMemcpy(phd_a, pd_a , sizeof(float *), hipMemcpyDeviceToHost);
        hipMemcpy(*pa, *phd_a , m * n * sizeof(float), hipMemcpyDeviceToHost);
    }
}

fdat *importData(void){
    fdat *dat = new fdat;
    FILE *datfile = fopen("externaltools/config","r");

    char *buffer = 0;
    long length;

    fseek (datfile, 0, SEEK_END);
    length = ftell (datfile);
    fseek (datfile, 0, SEEK_SET);
    buffer = (char *)malloc (length + 1);
    fread (buffer, 1, length, datfile);
    buffer[length] = '\0';

    fclose(datfile);

    if (buffer){
        DynamicJsonBuffer jsonBuffer;
        JsonObject& root = jsonBuffer.parseObject(buffer);
        if (!root.success()){
            printf("parseObject() failed\n");
        }
        else{
            dat->nx = root["nx"];
            dat->nz = root["nz"];
            dat->nt = root["nt"];
            dat->dt = root["dt"];
            dat->Lx = root["Lx"];
            dat->Lz = root["Lz"];

            dat->sfe = root["sfe"];
            dat->model_type = root["model_type"];
            dat->source_amplitude = root["source_amplitude"];

            if(root["src_info"].is<JsonObject>()){
                JsonObject& src = root["src_info"];
                DynamicJsonBuffer jsonBufferSrc;
                JsonArray& src_info = jsonBufferSrc.createArray();
                src_info.add(src);
                root.set("src_info",src_info);
            }

            JsonArray& src_info = root["src_info"];
            dat->nsrc = src_info.size();
            dat->stf_type = mat::createIntHost(dat->nsrc * sizeof(int));
            dat->src_x = mat::createHost(dat->nsrc * sizeof(float));
            dat->src_z = mat::createHost(dat->nsrc * sizeof(float));
            dat->stf_PSV_x = mat::createHost(dat->nsrc * sizeof(float));
            dat->stf_PSV_z = mat::createHost(dat->nsrc * sizeof(float));
            dat->tauw_0 = mat::createHost(dat->nsrc * sizeof(float));
            dat->tauw = mat::createHost(dat->nsrc * sizeof(float));
            dat->tee_0 = mat::createHost(dat->nsrc * sizeof(float));
            dat->f_min = mat::createHost(dat->nsrc * sizeof(float));
            dat->f_max = mat::createHost(dat->nsrc * sizeof(float));

            for(int i = 0; i < dat->nsrc; i++){
                JsonObject& src = src_info.get<JsonObject>(i);
                dat->src_x[i] = src["loc_x"];
                dat->src_z[i] = src["loc_z"];
                dat->stf_type[i] = 2; // ricker: modify later
                dat->stf_PSV_x[i] = src["stf_PSV"][0];
                dat->stf_PSV_z[i] = src["stf_PSV"][1];
                dat->tauw_0[i] = src["tauw_0"];
                dat->tauw[i] = src["tauw"];
                dat->tee_0[i] = src["tee_0"];
                dat->f_min[i] = src["f_min"];
                dat->f_max[i] = src["f_max"];
            }
        }
    }
    return dat;
}
float *importData(char *path, int *len){
    char fpath[50] = "externaltools/";
    strcat(fpath, path);
    *len = 0;
    float *data = 0;
    FILE *datafile = fopen(fpath,"r");
    if(datafile){
        while(!feof(datafile)){
            float datavalue;
            fscanf(datafile, "%f\n", &datavalue);
            *len = *len + 1;
        }
        fclose(datafile);

        datafile = fopen(fpath,"r");
        data = mat::createHost(*len);
        for(int i=0; i<*len; i++){
            fscanf(datafile, "%f\n", data + i);
        }
        fclose(datafile);
    }
    return data;
}
void exportData(float *data, int len, char *fname){
    char buffer[50] = "externaltools/";
    strcat(buffer, fname);
    FILE *file = fopen(buffer, "w");
    for(int i = 0; i < len; i++){
        fprintf(file, "%f\n", data[i]);
    }
    fclose(file);
}
void defineComputationalDomain(fdat *dat){
    dat->dx = dat->Lx / (dat->nx - 1);
    dat->dz = dat->Lz / (dat->nz - 1);
}
float *makeSourceTimeFunction(fdat *dat, int index){
    float *stf = mat::createHost(dat->nt);
    float max = 0;
    float alfa = 2 * dat->tauw_0[index] / dat->tauw[index];
    for(int i = 0; i < dat->nt; i++){
        float t = i * dat->dt;
        switch(dat -> stf_type[index]){
            case 2:{
                stf[i] = (-2 * pow(alfa, 3) / pi) * (t - dat->tee_0[index]) * exp(-pow(alfa, 2) * pow(t - dat->tee_0[index], 2));
                break;
            }
            // other stf: modify later
        }

        if(fabs(stf[i]) > max){
            max = fabs(stf[i]);
        }
    }
    if(max > 0){
        for(int i = 0; i < dat->nt; i++){
            stf[i] /= max;
        }
    }
    return stf;
}
void prepareSTF(fdat *dat){
    float *t = mat::createHost(dat->nt);
    int nt = dat->nt;
    for(int i = 0; i < nt; i++){
        t[i] = i * dat->dt;
    }

    dat->stf_x = mat::createHost(dat->nsrc, nt);
    dat->stf_y = mat::createHost(dat->nsrc, nt);
    dat->stf_z = mat::createHost(dat->nsrc, nt);
    float amp = dat->source_amplitude / dat->dx / dat->dz;
    for(int i=0; i < dat->nsrc; i++){
        float *stfn = makeSourceTimeFunction(dat, i);
        float px = dat->stf_PSV_x[i];
        float pz = dat->stf_PSV_z[i];
        float norm = sqrt(pow(px,2) + pow(pz,2));
        for(int j = 0; j < nt; j++){
            dat->stf_x[i][j] = amp * stfn[j] * px / norm;
            dat->stf_y[i][j] = amp * stfn[j];
            dat->stf_z[i][j] = amp * stfn[j] * pz / norm;
        }
    }
}
void checkArgs(fdat *dat){
    // int len;
    // add input file option: modify later
    // if update_params == 1  defineMaterialParameters here
    // float *stfall = importData("stf", &len);
    // if(len > 0){
    //     stf.stf_x = stfall;
    //     return stf;
    // }
    dat->update_params = 0;
    prepareSTF(dat);
}
void defineMaterialParameters(fdat *dat){
    // more model_type: modify later
    // int nx = dat->nx;
    // int nz = dat->nz;
    // switch(dat->model_type){
    //     case 1:{
    //         dat->rho = mat::init(mat::createHost(nx, nz), nx, nz, 3000);
    //         dat->mu = mat::init(mat::createHost(nx, nz), nx, nz, 4.8e10);
    //         dat->lambda = mat::init(mat::createHost(nx, nz), nx, nz, 4.8e10);
    //         break;
    //     }
    //     case 10:{
    //         dat->rho = mat::init(mat::createHost(nx, nz), nx, nz 2600);
    //         dat->mu = mat::init(mat::createHost(nx, nz), nx, nz 2.66e10);
    //         dat->lambda = mat::init(mat::createHost(nx, nz), nx, nz 3.42e10);
    //         break;
    //     }
    // }

}
void runWaveFieldPropagation(void){

}
void runForward(void){
    printf("initialising...\n");
    fdat *dat = importData();
    defineComputationalDomain(dat);
    checkArgs(dat);
    exportData(dat->stf_z[1],dat->nt,"stf_z"); // modify later
    if(!dat->update_params){
        defineMaterialParameters(dat);
    }

}

int main(int argc , char *argv[]){
    for(int i = 0; i< argc; i++){
        if(strcmp(argv[i],"runForward") == 0){
            runForward();
        }
    }

    return 0;
}
